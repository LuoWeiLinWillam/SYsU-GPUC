
#include <hip/hip_runtime.h>
#include<stdio.h>

void __global__ say_hello() {
  printf("Hello from thread %d\n", threadIdx.x);
}

int
main()
{
  say_hello<<<1, 10>>>();
  hipDeviceSynchronize();
  return 0;
}
